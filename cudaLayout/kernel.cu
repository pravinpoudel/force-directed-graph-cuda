#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>;
#include <>;
#include<iostream>
#include<stdio.h>
#include<cmath>
#include<vector>

using namespace std;


struct NodeLayout {
	int index = 0;
	struct Position {
		float x = rand()%10;
		float y = rand()%10;
	}position;

	struct Displacement {
		float x = 0.0;
		float y = 0.0;
	}displacement;
};

struct EdgeLayout {
	NodeLayout& node1;
	NodeLayout& node2;
	float weight;
};

typedef vector<NodeLayout> nodelistType;

struct GraphLayout {

	vector<NodeLayout> nodeList;
	vector<EdgeLayout> edgeList;

	void addNode(size_t node_count);
	void addEdge(size_t v0, size_t v1, float weight);
	void start(size_t iteration, int n_count);

};


void GraphLayout::addNode(size_t node_count) {
	for (int i = 0; i < node_count; i++) {
		NodeLayout node;
		node.index = i;
		printf("x %f and y %f\n", node.position.x, node.position.y);
		nodeList.push_back(node);
	}
}

void GraphLayout::addEdge(size_t v0, size_t v1, float weight) {
	if (v0 == v1 || weight == 0.0f || (nodeList.size() < max(v0, v1))) {
		return;
	}
	//please check why other initialization method not working; learn struct more in c++
	//EdgeLayout NewEdge;

	struct EdgeLayout NewEdge = { nodeList[v0], nodeList[v1], weight };
	edgeList.push_back(NewEdge);
}

__global__ void repulsiveForce(NodeLayout* nodeLayout, NodeLayout* currentNode, int n_count, float kSquare){
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < n_count) {
		float dx = nodeLayout[threadId].position.x - currentNode->position.x;
		float dy = nodeLayout[threadId].position.y - currentNode->position.y;
		if (dx && dy) {
			float d = dx * dx + dy * dy;
			float fr = (kSquare / sqrt(d));
			float cofficientx = dx / sqrt(d);
			float cofficienty = dy / sqrt(d);
			nodeLayout[threadId].displacement.x += fr * cofficientx;
			nodeLayout[threadId].displacement.y += fr * cofficienty;
		}
	}
}



void GraphLayout::start(size_t max_iteration_count, int n_count) {
	size_t nodeCount = nodeList.size();

	int WIDTH = 600;
	int HEIGHT = 400;
	float area = WIDTH * HEIGHT;
	float temperature = WIDTH / 10.0f;
	//optimal edge/link length 
	float k = sqrt(area / nodeCount);
	k = 10.0;
	float kSquare = area / nodeCount;
	kSquare = 100.0;
	
	int BLOCK_SIZE = 16;
	int GRID_SIZE = ceil((1.0f *n_count) / BLOCK_SIZE);

	NodeLayout* nodeLayout_Device;
	NodeLayout* ResultNodeLayout_Device;
	NodeLayout* currentNode_Device;

	int nodeListSize = sizeof(NodeLayout)*n_count;
	int currentNodeSize = sizeof(NodeLayout);
	
	hipMalloc((void**)&currentNode_Device, currentNodeSize);
	hipMalloc((void**)&nodeLayout_Device, nodeListSize);
	hipMalloc((void**)&ResultNodeLayout_Device, nodeListSize);

	int iterationCount = 0;

	while (iterationCount < max_iteration_count && temperature>0.00001f) {
		temperature *= (1.0 - ((iterationCount * 1.0) / max_iteration_count));
		iterationCount++;

		for (int i=0; i<n_count; i++) {
			nodeList[i].displacement = { 0.0f, 0.0f };
			hipMemcpy(nodeLayout_Device, &nodeList[0], nodeListSize, hipMemcpyHostToDevice);
			hipMemcpy(currentNode_Device, &nodeList[i], currentNodeSize, hipMemcpyHostToDevice);
			repulsiveForce <<< GRID_SIZE, BLOCK_SIZE >> > (nodeLayout_Device, currentNode_Device, n_count, kSquare);
			hipMemcpy(&nodeList[0], nodeLayout_Device, nodeListSize, hipMemcpyDeviceToHost);
		}



		for (auto iterator = edgeList.begin(); iterator != edgeList.end(); iterator++) {
			float dx = iterator->node1.position.x - iterator->node2.position.x;
			float dy = iterator->node1.position.y - iterator->node2.position.y;
			if (dx && dy) {
				float dSquare = dx * dx + dy * dy;
				float d = sqrt(dSquare);
				float fa = (dSquare / k);
				iterator->node1.displacement.x -= (dx / d) * fa;
				iterator->node1.displacement.y -= (dy / d) * fa;
				iterator->node2.displacement.x += (dx / d) * fa;
				iterator->node2.displacement.y += (dy / d) * fa;
			}

		}

		//limit displacement to the temperature
		for (auto iterator = nodeList.begin(); iterator != nodeList.end(); iterator++) {
			float d = sqrt(iterator->displacement.x * iterator->displacement.x + iterator->displacement.y * iterator->displacement.y);
			iterator->position.x += ((iterator->displacement.x) / d) * min(d, temperature);
			iterator->position.y += (iterator->displacement.y / d) * min(d, temperature);
		}
	}
}

int main() {
	GraphLayout graph;
	graph.addNode(100);
	graph.addEdge(0, 1, 1.0);
	graph.addEdge(0, 2, 1.0);
	graph.addEdge(1, 3, 1.0);
	graph.addEdge(2, 3, 1.0);
	graph.addEdge(3, 4, 1.0);
	graph.addEdge(1, 4, 1.0);
	graph.start(1000, 5);
	graph.addEdge(0, 2, 1.0);

	for (auto iterator = graph.nodeList.begin(); iterator != graph.nodeList.end(); iterator++) {
		printf(" node %d  coordinate is ( %f, %f)\n", iterator->index, iterator->position.x, iterator->position.y);
	}

}

